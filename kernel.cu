#include "hip/hip_runtime.h"
#include <cassert>                    // Biblioteca para realizar asserções
#include <hip/hip_runtime.h>             // Biblioteca CUDA para operações de tempo de execução
#include <> // Parâmetros de lançamento de dispositivo CUDA
#include "useGPU.h"                   // Inclui a definição da classe useGPU

using namespace std; // Espaço de nomes padrão do C++ (std)

// Função genérica para processar uma imagem no dispositivo GPU com uma função de kernel
template <typename Function>
void ProcessImageGpu(unsigned char *imageRGBA, int width, int height, Function kernelFunction)
{
    // Aloca memória no dispositivo para a imagem RGBA
    unsigned char *ptrImageDataGpu = nullptr;
    assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);                                   // Aloca memória
    assert(hipMemcpy(ptrImageDataGpu, imageRGBA, width * height * 4, hipMemcpyHostToDevice) == hipSuccess); // Copia dados para o dispositivo

    // Define o tamanho do bloco e da grade para paralelismo
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Chama a função de kernel no dispositivo
    kernelFunction<<<gridSize, blockSize>>>(ptrImageDataGpu);

    // Verifica se ocorreu algum erro no lançamento do kernel
    auto err = hipGetLastError();

    // Copia os dados de volta do dispositivo para a CPU
    assert(hipMemcpy(imageRGBA, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);

    // Libera a memória alocada no dispositivo
    hipFree(ptrImageDataGpu);
}

// Função de kernel para converter uma imagem colorida em escala de cinza
__global__ void ImageToGray(unsigned char *imageRGBA)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * gridDim.x * blockDim.x + x;

    unsigned char *pixel = &imageRGBA[idx * 4];
    unsigned char grayValue = static_cast<unsigned char>(0.2989 * pixel[0] + 0.5870 * pixel[1] + 0.1140 * pixel[2]);
    pixel[0] = grayValue;
    pixel[1] = grayValue;
    pixel[2] = grayValue;
}

// Função de kernel para realçar a cor vermelha na imagem
__global__ void ImageToRed(unsigned char *imageRGBA)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * gridDim.x * blockDim.x + x;

    unsigned char *pixel = &imageRGBA[idx * 4];
    pixel[1] = static_cast<unsigned char>(pixel[1] * 0.35);
    pixel[2] = static_cast<unsigned char>(pixel[2] * 0.35);
}

// Função de kernel para criar uma imagem negativa
__global__ void ImageToNegative(unsigned char *imageRGBA)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * gridDim.x * blockDim.x + x;

    unsigned char *pixel = &imageRGBA[idx * 4];
    pixel[0] = static_cast<unsigned char>(255 - pixel[0]);
    pixel[1] = static_cast<unsigned char>(255 - pixel[1]);
    pixel[2] = static_cast<unsigned char>(255 - pixel[2]);
}

// Método para converter uma imagem em escala de cinza no dispositivo GPU
void useGPU::ConvertImageToGrayGpu(unsigned char *imageRGBA, int width, int height)
{
    ProcessImageGpu(imageRGBA, width, height, ImageToGray);
}

// Método para realçar a cor vermelha na imagem no dispositivo GPU
void useGPU::ConvertImageToRedGpu(unsigned char *imageRGBA, int width, int height)
{
    ProcessImageGpu(imageRGBA, width, height, ImageToRed);
}

// Método para criar uma imagem negativa no dispositivo GPU
void useGPU::ConvertImageToNegativeGpu(unsigned char *imageRGBA, int width, int height)
{
    ProcessImageGpu(imageRGBA, width, height, ImageToNegative);
}
